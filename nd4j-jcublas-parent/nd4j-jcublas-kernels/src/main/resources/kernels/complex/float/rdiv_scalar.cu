extern "C"

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__global__ void rdiv_scalar_float(int n, int idx,float dx,float *dy,int incy,float *result) {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
                        if(i>= idx && i % incy == 0)
                             result[i] = dx / dy[i];
         }

 }


